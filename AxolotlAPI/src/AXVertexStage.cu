#include "hip/hip_runtime.h"
#include<pch.h>
#include"AXVertexStage.cuh"
#include"AXDeviceMemoryPool.cuh"
#include"AXInputAssembler.cuh"
#include"AXInputLayout.cuh"
#include"AXBuffer.cuh"
#include"AXFormat.h"

__global__ void KernelVertexProcess_Sample(void* vertices, unsigned int vertexCount, unsigned int vertexStride, void* outPtr, std::vector<AX_INPUT_ELEMENT>* input, std::vector<AX_INPUT_ELEMENT>* output)
{
	// do compute based on vertex shader.
	// output structure must be determined before the vertex stage. (IA stage)


	unsigned int vertexOffset = threadIdx.x * vertexStride;
	void* vertex = (int*)vertices + vertexOffset;

	unsigned int inputCount = input->size();

	std::vector<void*> inputElements(inputCount);
	void* buffer;
	for (unsigned int i = 0; i < inputCount; i++)
	{
		void* offsetElement = (size_t*)vertex + (*input)[i].Offset;
		
		AX_PIXEL_DESC pixDesc = GetPixelDesc((*input)[i].Format);
		unsigned int elementByteSize = (pixDesc.Components * pixDesc.BitPerComponent) / 8;

		inputElements.push_back(offsetElement);

		// update from here 2021/12/28 11:38 AM

	}

	unsigned int outputCount = output->size();
	void* out = (int*)outPtr + vertexOffset;
	for (unsigned int i = 0; i < outputCount; i++)
	{
		void* offsetElement = (size_t*)outPtr + (*output)[i].Offset;

		unsigned int slotId = (*output)[i].Slot;
		
		
	}


	

	return;
}

AXVertexStage::AXVertexStage(std::shared_ptr<AXDeviceMemoryPool> memory)
{
	//mInputElements = memory->Alloc<std::vector<AX_INPUT_ELEMENT>>(4096);
	//mOutputElements = memory->Alloc<std::vector<AX_INPUT_ELEMENT>>(4096);
	// --------------------------------------------------------

	AX_INPUT_ELEMENT inputStructure[] =
	{
		{"POSITION", 0, 0,0, eAXFormat::AX_R32G32B32_FLOAT},
		{"COLOR", 1, 0, 12, eAXFormat::AX_R32G32B32A32_FLOAT},
	};

	AX_INPUT_ELEMENT outputStructure[] =
	{
		{"SV_Position", 0, 0, 0, eAXFormat::AX_R32G32B32A32_FLOAT },
	};


	mHostOutputElements.push_back(outputStructure[0]);
	
}

AXVertexStage::~AXVertexStage()
{
}

void AXVertexStage::Process(std::shared_ptr<AXInputAssembler> assembler)
{
	unsigned int vbCount = assembler->VertexBuffers.size();

	for (unsigned int i = 0; i < vbCount; i++)
	{
		std::shared_ptr<AXBuffer> buffer = assembler->VertexBuffers[i];

		void* virtualAddr = buffer->GetVirtual();

		AX_BUFFER_DESC bufferDesc = buffer->GetDesc();
		
		unsigned int vertexStride = assembler->VertexStrides[i];
		unsigned int vertexCount = bufferDesc.ByteSize / assembler->VertexStrides[i];
		
		dim3 grid = dim3(vertexCount);
		dim3 block = dim3(32);

		std::vector<AX_INPUT_ELEMENT>* inputElements = &assembler->InputLayout->mElements;
		std::vector<AX_INPUT_ELEMENT>* outputElements = &mHostOutputElements;
		hipMemcpy(mInputElements, inputElements, inputElements->size() * sizeof(AX_INPUT_ELEMENT), hipMemcpyHostToDevice);
		hipMemcpy(mOutputElements, outputElements, outputElements->size() * sizeof(AX_INPUT_ELEMENT), hipMemcpyHostToDevice);

		KernelVertexProcess_Sample <<<grid, block>>>(virtualAddr, vertexCount, vertexStride, mVertexOutput, gInputStructure, gOutputStructure);
		hipDeviceSynchronize();
	}
}
