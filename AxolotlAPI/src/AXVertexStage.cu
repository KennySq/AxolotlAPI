#include "hip/hip_runtime.h"
#include<pch.h>
#include"AXVertexStage.cuh"
#include"AXInputAssembler.cuh"
#include"AXInputLayout.cuh"
#include"AXBuffer.cuh"
#include"AXFormat.h"

__device__ std::vector<AX_INPUT_ELEMENT>* gInputStructure;
__device__ std::vector<AX_INPUT_ELEMENT>* gOutputStructure;

__global__ void KernelVertexProcess_Sample(void* vertices, unsigned int vertexCount, unsigned int vertexStride, void* outPtr, std::vector<AX_INPUT_ELEMENT>* input, std::vector<AX_INPUT_ELEMENT>* output)
{
	// do compute based on vertex shader.
	// output structure must be determined before the vertex stage. (IA stage)

	unsigned int vertexOffset = threadIdx.x * vertexStride;
	void* vertex = (int*)vertices + vertexOffset;
	
	unsigned int inputCount = input->size();

	std::vector<void*> inputElements(inputCount);
	void* buffer;
	for (unsigned int i = 0; i < inputCount; i++)
	{
		void* offsetElement = (size_t*)vertex + (*input)[i].Offset;
		
		AX_PIXEL_DESC pixDesc = GetPixelDesc((*input)[i].Format);
		unsigned int elementByteSize = (pixDesc.Components * pixDesc.BitPerComponent) / 8;

		inputElements.push_back(offsetElement);

		// update from here 2021/12/28 11:38 AM

	}

	unsigned int outputCount = output->size();
	void* out = (int*)outPtr + vertexOffset;
	for (unsigned int i = 0; i < outputCount; i++)
	{
		void* offsetElement = (size_t*)outPtr + (*output)[i].Offset;

		unsigned int slotId = (*output)[i].Slot;

	}


	

	return;
}

AXVertexStage::AXVertexStage()
{
	hipMalloc(reinterpret_cast<void**>(&gInputStructure), 4096);
	hipMalloc(reinterpret_cast<void**>(&gOutputStructure), 4096);

	// --------------------------------------------------------

	AX_INPUT_ELEMENT outputElements[] =
	{
		{"SV_Position", 0, 0, 0, eAXFormat::AX_R32G32B32A32_FLOAT },
	};
	mOutputElements.push_back(outputElements[0]);
}

AXVertexStage::~AXVertexStage()
{
}

void AXVertexStage::Process(std::shared_ptr<AXInputAssembler> assembler)
{
	unsigned int vbCount = assembler->VertexBuffers.size();

	for (unsigned int i = 0; i < vbCount; i++)
	{
		std::shared_ptr<AXBuffer> buffer = assembler->VertexBuffers[i];

		void* virtualAddr = buffer->GetVirtual();

		AX_BUFFER_DESC bufferDesc = buffer->GetDesc();
		
		unsigned int vertexStride = assembler->VertexStrides[i];
		unsigned int vertexCount = bufferDesc.ByteSize / assembler->VertexStrides[i];
		
		dim3 grid = dim3(vertexCount);
		dim3 block = dim3(32);

		std::vector<AX_INPUT_ELEMENT>* inputElements = &assembler->InputLayout->mElements;
		std::vector<AX_INPUT_ELEMENT>* outputElements = &mOutputElements;
		hipMemcpy(gInputStructure, inputElements, inputElements->size() * sizeof(AX_INPUT_ELEMENT), hipMemcpyHostToDevice);
		hipMemcpy(gOutputStructure, outputElements, outputElements->size() * sizeof(AX_INPUT_ELEMENT), hipMemcpyHostToDevice);

		KernelVertexProcess_Sample <<<grid, block>>>(virtualAddr, vertexCount, vertexStride, mVertexOutput, gInputStructure, gOutputStructure);
		hipDeviceSynchronize();
	}
}
