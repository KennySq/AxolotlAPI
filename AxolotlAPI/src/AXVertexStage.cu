#include "hip/hip_runtime.h"
#include<pch.h>
#include"AXVertexStage.cuh"
#include"AXInputAssembler.cuh"
#include"AXBuffer.cuh"

__global__ void KernelVertexProcess(void* vertices, unsigned int vertexCount, void* outPtr)
{
	// do compute based on vertex shader.
	// output structure must be determined before vertex stage. (IA stage)


	return;
}

AXVertexStage::AXVertexStage()
{
}

AXVertexStage::~AXVertexStage()
{
}

void AXVertexStage::Process(std::shared_ptr<AXInputAssembler> assembler)
{
	unsigned int vbCount = assembler->VertexBuffers.size();
	for (unsigned int i = 0; i < vbCount; i++)
	{
		std::shared_ptr<AXBuffer> buffer = assembler->VertexBuffers[i];

		void* virtualAddr = buffer->GetVirtual();

		AX_BUFFER_DESC bufferDesc = buffer->GetDesc();
		unsigned int vertexCount = bufferDesc.ByteSize / assembler->VertexStrides[i];

		dim3 grid = dim3(vertexCount);
		dim3 block = dim3(32);
		KernelVertexProcess<<<grid, block>>>(virtualAddr, vertexCount, mVertexOutput);
	}
}
