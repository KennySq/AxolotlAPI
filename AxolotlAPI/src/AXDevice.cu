#include "hip/hip_runtime.h"
#include"pch.h"
#include "AXDevice.cuh"
#include"AXTexture.h"
#include"AXCommandList.h"
#include"AXRenderTargetView.cuh"
#include"AXDeviceTexture2D.cuh"
#include"AXDeviceMemoryPool.cuh"

#include<Windows.h>

unsigned int AXDevice::mInterfaceCounter = 0;

static __global__ void KernelWriteRaw(int data, void* raw, unsigned int offset)
{
	void* ptr = (void*)((size_t)raw + offset);
	int* asInt = reinterpret_cast<int*>(ptr);

	*asInt = data;
}

static __global__ void KernelViewRaw(unsigned int offset, void* raw)
{
	void* ptr = (void*)((size_t)raw + offset);
	int* asInt = reinterpret_cast<int*>(ptr);

	printf("%d\n", *reinterpret_cast<int*>(&asInt[0]));
	printf("%d\n", *reinterpret_cast<int*>(&asInt[1]));
	printf("%d\n", *reinterpret_cast<int*>(&asInt[2]));
}

AXDevice::AXDevice(unsigned int flag)
	: mDeviceFlag(flag)
{
	mMemory = std::make_shared<AXDeviceMemoryPool>(8294400);
	KernelViewRaw<<<1,1>>>(0, pool.GetRaw());
	hipDeviceSynchronize();
}

AXDevice::~AXDevice()
{
}

std::shared_ptr<AXTexture2D> AXDevice::CreateTexture2D(const AX_TEXTURE2D_DESC& desc)
{
	unsigned int perPixel = 0;
	unsigned int totalSize = 0;
	if (desc.Format == AX_R8G8B8A8_FLOAT)
	{
		perPixel = 32;
		totalSize = desc.Width * desc.Height * perPixel;
	}

	void* devicePtr = mMemory->Alloc(totalSize);

	std::shared_ptr<AXTexture2D> tex = std::make_shared<AXTexture2D>();
	// update from here 2021/12/16 2:51 PM
	CreateDeviceTexture2D << <1, 1, 1 >> > (desc);
	tex->mWidth = desc.Width;
	tex->mHeight = desc.Height;
	tex->mFormat = desc.Format;
	
	mInterfaceCounter++;

	return tex;
}

std::shared_ptr<AXCommandList> AXDevice::CreateCommandList()
{
	std::shared_ptr<AXCommandList> cmdList = std::make_shared<AXCommandList>();
	
	cmdList->mCommands.resize(1024);

	mInterfaceCounter++;

	return cmdList;
}

std::shared_ptr<AXDevice> AXCreateDevice(unsigned int flag)
{
	std::shared_ptr<AXDevice> device = std::make_shared<AXDevice>(flag);

	if (flag & CREATE_AXDEVICE_DEBUG)
	{
		std::cout << "Debug Layer Active.\n";
		OutputDebugStringA("Debug Layer Active.\n");
	}



	return device;
}


std::shared_ptr<AXRenderTargetView> AXDevice::CreateRenderTargetView(const AX_RENDER_TARGET_VIEW_DESC& desc)
{
	std::shared_ptr<AXRenderTargetView> rtv = std::make_shared<AXRenderTargetView>();

	rtv->mDimension = desc.Dimension;

	return rtv;
}