#include "hip/hip_runtime.h"
#include"pch.h"
#include "AXDevice.cuh"
#include"AXTexture.h"
#include"AXCommandList.h"
#include"AXRenderTargetView.cuh"
#include"AXDeviceTexture2D.cuh"
#include"AXDeviceMemoryPool.cuh"
#include"AXBuffer.cuh"

#include"IAXResource.h"

#include<Windows.h>

unsigned int AXDevice::mInterfaceCounter = 0;

__declspec(dllexport) __global__ void KernelWriteRaw(int data, void* raw, unsigned int offset)
{
	void* ptr = (void*)((size_t)raw + offset);
	int* asInt = reinterpret_cast<int*>(ptr);

	*asInt = data;
}

__declspec(dllexport) __global__ void KernelViewRaw(unsigned int offset, void* raw)
{
	void* ptr = (void*)((size_t)raw + offset);
	int* asInt = reinterpret_cast<int*>(ptr);

	printf("%d\n", *reinterpret_cast<int*>(&asInt[0]));
	printf("%d\n", *reinterpret_cast<int*>(&asInt[1]));
	printf("%d\n", *reinterpret_cast<int*>(&asInt[2]));
}

AXDevice::AXDevice(unsigned int flag)
	: mDeviceFlag(flag)
{
	mMemory = std::make_shared<AXDeviceMemoryPool>(8294400);
	//KernelWriteRaw << <1, 1, 1 >> > (162, mMemory->GetRaw(), 0);
	//hipDeviceSynchronize();

	//KernelViewRaw<<<1,1,1>>>(0, mMemory->GetRaw());
	//hipDeviceSynchronize();
}

AXDevice::~AXDevice()
{
	mMemory->ReleasePool();
}

std::shared_ptr<AXTexture2D> AXDevice::CreateTexture2D(const AX_TEXTURE2D_DESC& desc)
{
	unsigned int perPixel = 0;
	unsigned int totalSize = 0;

	AX_PIXEL_DESC pixelDesc = GetPixelDesc(desc.Format);

	totalSize = (desc.Width * desc.Height * pixelDesc.BitPerComponent * pixelDesc.Components) / 8;

	void* devicePtr = mMemory->Alloc(totalSize);

	std::shared_ptr<AXTexture2D> tex = std::make_shared<AXTexture2D>();
	// update from here 2021/12/16 2:51 PM

	tex->mWidth = desc.Width;
	tex->mHeight = desc.Height;
	tex->mFormat = desc.Format;
	tex->mRaw = devicePtr;
	
	mInterfaceCounter++;

	return tex;
}

std::shared_ptr<AXCommandList> AXDevice::CreateCommandList()
{
	std::shared_ptr<AXCommandList> cmdList = std::make_shared<AXCommandList>();
	
	cmdList->mCommands.resize(1024);

	mInterfaceCounter++;

	return cmdList;
}

std::shared_ptr<AXBuffer> AXDevice::CreateBuffer(const AX_BUFFER_DESC& desc)
{
	std::shared_ptr<AXBuffer> buffer = std::make_shared<AXBuffer>();

	buffer->mRaw = mMemory->Alloc(desc.ByteSize);
	buffer->mSize = desc.ByteSize;
	buffer->mBindFlags = desc.BindFlags;

	return buffer;
}

std::shared_ptr<AXDevice> AXCreateDevice(unsigned int flag)
{
	std::shared_ptr<AXDevice> device = std::make_shared<AXDevice>(flag);

	if (flag & CREATE_AXDEVICE_DEBUG)
	{
		std::cout << "Debug Layer Active.\n";
		OutputDebugStringA("Debug Layer Active.\n");
	}

	return device;
}


std::shared_ptr<AXRenderTargetView> AXDevice::CreateRenderTargetView(std::shared_ptr<IAXResource> resource, const AX_RENDER_TARGET_VIEW_DESC& desc)
{
	std::shared_ptr<AXRenderTargetView> rtv = std::make_shared<AXRenderTargetView>();

	rtv->mDimension = desc.Dimension;
	rtv->mResource = resource;

	return rtv;
}