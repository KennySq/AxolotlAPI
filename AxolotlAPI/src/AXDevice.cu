#include "hip/hip_runtime.h"
#include"pch.h"
#include "AXDevice.cuh"
#include"AXTexture.h"
#include"AXCommandList.h"
#include"AXRenderTargetView.cuh"
#include"AXDeviceTexture2D.cuh"
#include"AXDeviceMemoryPool.cuh"
#include"AXBuffer.cuh"
#include"AXInputLayout.cuh"

#include"AXVertexShader.cuh"

#include"AXBytecode.h"


#include"IAXResource.h"

#include<Windows.h>

unsigned int AXDevice::mInterfaceCounter = 0;

__declspec(dllexport) __global__ void KernelWriteRaw(int data, void* raw, unsigned int offset)
{
	void* ptr = (void*)((size_t)raw + offset);
	int* asInt = reinterpret_cast<int*>(ptr);

	*asInt = data;
}

__declspec(dllexport) __global__ void KernelViewRaw(unsigned int offset, void* raw)
{
	void* ptr = (void*)((size_t)raw + offset);
	int* asInt = reinterpret_cast<int*>(ptr);

	printf("%d\n", *reinterpret_cast<int*>(&asInt[0]));
	printf("%d\n", *reinterpret_cast<int*>(&asInt[1]));
	printf("%d\n", *reinterpret_cast<int*>(&asInt[2]));
}

AXDevice::AXDevice(unsigned int flag)
	: mDeviceFlag(flag)
{
	hipDeviceProp_t prop;
	int deviceIndex;

	hipDeviceSynchronize();

	hipGetDevice(&deviceIndex);
	hipGetDeviceProperties(&prop, deviceIndex);
	
	std::string memoryStr = std::to_string(prop.totalGlobalMem / 4) + " bytes";
	Log(std::string("Available device memory : ") + memoryStr);

	mMemory = std::make_shared<AXDeviceMemoryPool>(prop.totalGlobalMem / 4);
}

AXDevice::~AXDevice()
{
	mMemory->ReleasePool();
}

std::shared_ptr<AXTexture2D> AXDevice::CreateTexture2D(const AX_TEXTURE2D_DESC& desc)
{
	unsigned int perPixel = 0;
	unsigned int totalSize = 0;

	AX_PIXEL_DESC pixelDesc = GetPixelDesc(desc.Format);

	totalSize = (desc.Width * desc.Height * pixelDesc.BitPerComponent * pixelDesc.Components) / 8;

	void* devicePtr = mMemory->Alloc(totalSize);

	std::shared_ptr<AXTexture2D> tex = std::make_shared<AXTexture2D>();
	// update from here 2021/12/16 2:51 PM

	tex->mWidth = desc.Width;
	tex->mHeight = desc.Height;
	tex->mFormat = desc.Format;
	tex->mRaw = devicePtr;

	mInterfaceCounter++;

	return tex;
}

std::shared_ptr<AXCommandList> AXDevice::CreateCommandList()
{
	std::shared_ptr<AXCommandList> cmdList = std::make_shared<AXCommandList>();

	cmdList->mCommands.resize(1024);

	mInterfaceCounter++;

	return cmdList;
}

std::shared_ptr<AXInputLayout> AXDevice::CreateInputLayout(AX_INPUT_ELEMENT* const inputElements, unsigned int elementCount)
{
	std::shared_ptr<AXInputLayout> il = std::make_shared<AXInputLayout>();
	
	il->mElements.resize(elementCount);

	for (unsigned int i = 0; i < elementCount; i++)
	{	
		il->mElements[i] = inputElements[i];
	}

	return il;
}

std::shared_ptr<AXBuffer> AXDevice::CreateBuffer(const AX_BUFFER_DESC& desc, void* subResource)
{
	std::shared_ptr<AXBuffer> buffer = std::make_shared<AXBuffer>();

	buffer->mRaw = mMemory->Alloc(desc.ByteSize);

	if (subResource != nullptr)
	{
		hipError_t error = hipMemcpy(buffer->mRaw, subResource, desc.ByteSize, hipMemcpyHostToDevice);

		if (error != NULL)
		{
			Log("subresource copy fail");
			Log(hipGetErrorString(error));
		}
	}

	buffer->mSize = desc.ByteSize;
	buffer->mBindFlags = desc.BindFlags;

	return buffer;
}

std::shared_ptr<AXDevice> AXCreateDevice(unsigned int flag)
{
	std::shared_ptr<AXDevice> device = std::make_shared<AXDevice>(flag);

	if (flag & CREATE_AXDEVICE_DEBUG)
	{
		std::cout << "Debug Layer Active.\n";
		OutputDebugStringA("Debug Layer Active.\n");
	}

	return device;
}


std::shared_ptr<AXRenderTargetView> AXDevice::CreateRenderTargetView(std::shared_ptr<IAXResource> resource, const AX_RENDER_TARGET_VIEW_DESC& desc)
{
	std::shared_ptr<AXRenderTargetView> rtv = std::make_shared<AXRenderTargetView>();

	rtv->mDimension = desc.Dimension;
	rtv->mResource = resource;

	return rtv;
}

std::shared_ptr<AXVertexShader> AXDevice::CreateVertexShader(std::shared_ptr<AXBytecode> bytecode)
{
	//SlimShader::BytecodeReader reader = SlimShader::BytecodeReader(bytecode->)
	//return std::shared_ptr<AXVertexShader>();

	return nullptr;
}
