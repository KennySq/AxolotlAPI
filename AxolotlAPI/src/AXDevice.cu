#include "hip/hip_runtime.h"
#include"pch.h"
#include "AXDevice.cuh"
#include"AXTexture.h"
#include"AXCommandList.h"
#include"AXRenderTargetView.h"

#include<Windows.h>

unsigned int AXDevice::mInterfaceCounter = 0;

AXDevice::AXDevice(unsigned int flag)
	: mDeviceFlag(flag)
{
}

AXDevice::~AXDevice()
{
}

__global__ void KernelCreateTexture2D(const AX_TEXTURE2D_DESC desc)
{
	
}

std::shared_ptr<AXTexture2D> AXDevice::CreateTexture2D(const AX_TEXTURE2D_DESC& desc)
{

	mInterfaceCounter++;

	return nullptr;
}

std::shared_ptr<AXCommandList> AXDevice::CreateCommandList()
{
	std::shared_ptr<AXCommandList> cmdList = std::make_shared<AXCommandList>();
	
	cmdList->mCommands.resize(1024);

	mInterfaceCounter++;

	return cmdList;
}

std::shared_ptr<AXDevice> AXCreateDevice(unsigned int flag)
{
	std::shared_ptr<AXDevice> device = std::make_shared<AXDevice>(flag);

	if (flag & CREATE_AXDEVICE_DEBUG)
	{
		std::cout << "Debug Layer Active.\n";
		OutputDebugStringA("Debug Layer Active.\n");
	}



	return device;
}


std::shared_ptr<AXRenderTargetView> AXDevice::CreateRenderTargetView(const AX_RENDER_TARGET_VIEW_DESC& desc)
{
	std::shared_ptr<AXRenderTargetView> rtv = std::make_shared<AXRenderTargetView>();

	rtv->mDimension = desc.Dimension;

	return rtv;
}