#include "hip/hip_runtime.h"
#include "pch.h"
#include "AXContext.h"
#include"AXCommandList.h"
#include"Command.h"

#include"AXTexture.h"
#include"AXRenderTargetView.cuh"

AXContext::AXContext(unsigned int flag)
	: mFlag(flag), mCommandIndex(0), mCommandBuffer(1024)
{
}

AXContext::~AXContext()
{
}

__global__ void KernelClearRenderTarget(void* ptr, unsigned int width, unsigned height, unsigned int componentSize, float r, float g, float b, float a)
{
	unsigned int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	unsigned int index = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	
	DWORD* asPixel = reinterpret_cast<DWORD*>(ptr);

	BYTE comp0 = r * 255.999f;
	BYTE comp1 = g * 255.999f;
	BYTE comp2 = b * 255.999f;
	BYTE comp3 = a * 255.999f;

	DWORD color = 0;

	color |= (comp3 << 24);
	color |= (comp0 << 16);
	color |= (comp1 << 8);
	color |= (comp2 << 0);

	asPixel[index] = color;
}

void AXContext::ClearRenderTarget(std::shared_ptr<AXRenderTargetView> rtv, float clearColor[4])
{
	std::shared_ptr<IAXResource> resource = rtv->mResource;
	std::shared_ptr<AXTexture2D> asTex2d = std::static_pointer_cast<AXTexture2D>(resource);

	AX_TEXTURE2D_DESC desc = asTex2d->GetDesc();
	unsigned int width = desc.Width;
	unsigned int height = desc.Height;

	dim3 block = dim3(8, 8, 1);
	dim3 grid = dim3(width / block.y, height / block.y, 1);

	Command cmd;

	float r, g, b, a;

	r = clearColor[0];
	g = clearColor[1];
	b = clearColor[2];
	a = clearColor[3];

	KernelClearRenderTarget << <grid, block >> > (resource->mRaw, width, height, 8, r, g, b, a);
	hipDeviceSynchronize();
	//cmd.Bind<8, void*, unsigned int, unsigned int, unsigned int, float, float, float, float>
	//	(WrapperClearRenderTarget, resource->mRaw, width, height, (unsigned int)8, clearColor[0], clearColor[1], clearColor[2], clearColor[3]);

	//mCommandBuffer[mCommandIndex] = cmd;
	//mCommandIndex++;
	//cmd.Bind<void*, unsigned int, unsigned int, unsigned int, float, float, float, float>
	//	(WrapperClearRenderTarget, resource->mRaw, width, height, (unsigned int)8, clearColor[0], clearColor[1], clearColor[2], clearColor[3]);
	//// update from here 2021/12/16 10:38 PM
	//auto b = std::bind(WrapperClearRenderTarget, resource->mRaw, width, height, (unsigned int)8, clearColor[0], clearColor[1], clearColor[2], clearColor[3]);
	//std::bind<std::function<void(float)>, float>(floatFn, 0.0f);
	//mCommandBuffer.push_back();
}

void AXContext::ExecuteCommandList(const std::shared_ptr<AXCommandList>& cmdList) const
{
	unsigned int size = cmdList->mCommands.size();

	for (unsigned int i = 0; i < size; i++)
	{
		cmdList->mCommands[i].Execute();
	}

	cmdList->mbClosed = false;
}

void AXContext::FinishCommandList(std::shared_ptr<AXCommandList>* cmdList)
{
	unsigned int size = mCommandBuffer.size();

	for (unsigned int i = 0; i < size; i++)
	{
		Command cmd = mCommandBuffer[i];
		(*cmdList)->mCommands[i] = cmd;
	}
	(*cmdList)->mbClosed = true;

	mCommandBuffer.clear();
	mCommandBuffer.resize(1024);
	mCommandIndex = 0;
}
