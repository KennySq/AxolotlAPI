#include "hip/hip_runtime.h"
#include "pch.h"
#include "AXContext.h"
#include"AXCommandList.h"
#include"Command.h"

#include"AXTexture.h"
#include"AXRenderTargetView.cuh"

AXContext::AXContext(unsigned int flag)
	: mFlag(flag)
{
}

AXContext::~AXContext()
{
}

__global__ void KernelClearRenderTarget(void* ptr, unsigned int width, unsigned height, unsigned int componentSize, float r, float g, float b, float a)
{
	printf("Kernel Call, Clear Render Target View!\n");
}

void WrapperClearRenderTarget(void* ptr, unsigned int width, unsigned int height, unsigned int componentSize, float r, float g, float b, float a)
{

}

void AXContext::ClearRenderTarget(std::shared_ptr<AXRenderTargetView> rtv, float clearColor[4])
{
	std::shared_ptr<IAXResource> resource = rtv->mResource;

	unsigned int width = 1280;
	unsigned int height = 720;
	KernelClearRenderTarget<<<width, height, 1>>>(resource->mRaw, width, height, 8, clearColor[0], clearColor[1], clearColor[2], clearColor[3]);
	hipDeviceSynchronize();

	//Command cmd = Command(WrapperClearRenderTarget);
	// update from here 2021/12/16 10:38 PM

	mCommandBuffer.push_back();
}

void AXContext::ExecuteCommandList(const std::shared_ptr<AXCommandList>& cmdList) const
{
	unsigned int size = cmdList->mCommands.size();

	for (unsigned int i = 0; i < size; i++)
	{
		cmdList->mCommands[i].Execute();
	}

	cmdList->mbClosed = false;
}

void AXContext::FinishCommandList(std::shared_ptr<AXCommandList>* cmdList)
{
	unsigned int size = mCommandBuffer.size();

	for (unsigned int i = 0; i < size; i++)
	{
		Command cmd = mCommandBuffer[i];
		(*cmdList)->mCommands.push_back(cmd);
	}
	(*cmdList)->mbClosed = true;

	mCommandBuffer.clear();
}
