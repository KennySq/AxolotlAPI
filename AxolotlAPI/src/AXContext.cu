#include "hip/hip_runtime.h"
#include "pch.h"
#include "AXContext.h"
#include"AXUtil.h"
#include"AXCommandList.h"
#include"Command.h"

#include"AXInputAssembler.cuh"
#include"AXVertexStage.cuh"
#include"AXRasterizer.cuh"
#include"AXOutputMerger.cuh"
#include"AXPixelStage.cuh"

#include"AX3DMath.h"

#include"AXTexture.h"
#include"AXRenderTargetView.cuh"
#include"AXBuffer.cuh"

AXContext::AXContext(unsigned int flag)
	: mFlag(flag), mCommandIndex(0), mCommandBuffer(1024)
{
}

AXContext::~AXContext()
{
}

__global__ void KernelClearRenderTarget(void* ptr, unsigned int width, unsigned height, unsigned int componentSize, float r, float g, float b, float a)
{
	unsigned int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	unsigned int index = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	
	DWORD* asPixel = reinterpret_cast<DWORD*>(ptr);

	asPixel[index] = deviceConvertRGB(r, g, b, a);
}

void AXContext::IASetVertexBuffer(std::shared_ptr<AXBuffer>* const buffer, unsigned int count, unsigned int* const strides, unsigned int* const offsets)
{
	assert(strides != nullptr && "strides cannot be null.");

	for (unsigned int i = 0; i < count; i++)
	{
		AX_BIND_FLAG bindFlag = buffer[i]->mBindFlags;
		
		if ((bindFlag & AX_BIND_VERTEX_BUFFER) != 1)
		{
			Log("This buffer cannot be bound on IA stage.");
		}

		mAssembler->VertexBuffers.push_back(buffer[i]);
		mAssembler->VertexStrides.push_back(strides[i]);
	}
}

void AXContext::IASetIndexBuffer(std::shared_ptr<AXBuffer> buffer)
{
	if (buffer == nullptr)
	{
		Log("Invalid argument");
	}

	AX_BIND_FLAG bindFlag = buffer->mBindFlags;
	if ((bindFlag & AX_BIND_INDEX_BUFFER) != 1)
	{
		Log("This buffer cannot be bound on IA stage.");
	}

	mAssembler->IndexBuffer = buffer;


}

void AXContext::DrawIndexed(unsigned int indexCount, unsigned int offset)
{
	std::shared_ptr<AXRenderTargetView> rtv = mOutputStage->GetRenderTargetView(0);
	std::shared_ptr<IAXResource> resource = rtv->mResource;
	std::shared_ptr<AXTexture2D> asTex2d = std::static_pointer_cast<AXTexture2D>(resource);

	AX_TEXTURE2D_DESC texDesc = asTex2d->GetDesc();
	unsigned int vertexCount = mAssembler->VertexBuffers[0]->mSize / mAssembler->VertexStrides[0];

	mVertexStage->Process(mAssembler);
	hipDeviceSynchronize();
	mRasterStage->Process(resource, texDesc.Width, texDesc.Height, mVertexStage->GetOutput(), vertexCount);
	hipDeviceSynchronize();
}

void AXContext::ClearRenderTarget(std::shared_ptr<AXRenderTargetView> rtv, float clearColor[4])
{
	std::shared_ptr<IAXResource> resource = rtv->mResource;
	std::shared_ptr<AXTexture2D> asTex2d = std::static_pointer_cast<AXTexture2D>(resource);

	AX_TEXTURE2D_DESC desc = asTex2d->GetDesc();
	unsigned int width = desc.Width;
	unsigned int height = desc.Height;

	dim3 block = dim3(32, 30, 1);
	dim3 grid = dim3(width / block.y, height / block.y, 1);

	Command cmd;

	float r, g, b, a;

	r = clearColor[0];
	g = clearColor[1];
	b = clearColor[2];
	a = clearColor[3];

	KernelClearRenderTarget << <grid, block >> > (resource->mRaw, width, height, 8, r, g, b, a);
	hipDeviceSynchronize();
	//cmd.Bind<8, void*, unsigned int, unsigned int, unsigned int, float, float, float, float>
	//	(WrapperClearRenderTarget, resource->mRaw, width, height, (unsigned int)8, clearColor[0], clearColor[1], clearColor[2], clearColor[3]);

	//mCommandBuffer[mCommandIndex] = cmd;
	//mCommandIndex++;
	//cmd.Bind<void*, unsigned int, unsigned int, unsigned int, float, float, float, float>
	//	(WrapperClearRenderTarget, resource->mRaw, width, height, (unsigned int)8, clearColor[0], clearColor[1], clearColor[2], clearColor[3]);
	//// update from here 2021/12/16 10:38 PM
	//auto b = std::bind(WrapperClearRenderTarget, resource->mRaw, width, height, (unsigned int)8, clearColor[0], clearColor[1], clearColor[2], clearColor[3]);
	//std::bind<std::function<void(float)>, float>(floatFn, 0.0f);
	//mCommandBuffer.push_back();
}

void AXContext::ExecuteCommandList(const std::shared_ptr<AXCommandList>& cmdList) const
{
	unsigned int size = cmdList->mCommands.size();

	for (unsigned int i = 0; i < size; i++)
	{
		cmdList->mCommands[i].Execute();
	}

	cmdList->mbClosed = false;
}

void AXContext::FinishCommandList(std::shared_ptr<AXCommandList>* cmdList)
{
	unsigned int size = mCommandBuffer.size();

	for (unsigned int i = 0; i < size; i++)
	{
		Command cmd = mCommandBuffer[i];
		(*cmdList)->mCommands[i] = cmd;
	}
	(*cmdList)->mbClosed = true;

	mCommandBuffer.clear();
	mCommandBuffer.resize(1024);
	mCommandIndex = 0;
}
