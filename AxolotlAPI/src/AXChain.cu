#include<pch.h>
#include "AXChain.cuh"
#include"AXTexture.h"
#include"AXUtil.h"

AXChain::AXChain()
{


}

AXChain::~AXChain()
{
}

void AXChain::Present()
{
	unsigned int size = (mWidth * mHeight *  4);
	hipError_t error = hipMemcpy(mBuffer, mVirtual, size, hipMemcpyDeviceToHost);
	if (error != NULL)
	{
		Log(hipGetErrorString(error));
	}

	hipDeviceSynchronize();

	BitBlt(mMainDC, 0, 0, mWidth, mHeight, mMemoryDC, 0, 0, SRCCOPY);
}

void AXChain::BindTexture(unsigned int index, const std::shared_ptr<AXTexture2D>& outTexture)
{
	AX_TEXTURE2D_DESC texDesc{};

	//if (outTexture->mFormat != AX_R8G8B8A8_UNORM)
	//{
	//	Log("invalid format.");
	//	return;
	//}

	mVirtual = outTexture->mRaw;
}