#include "hip/hip_runtime.h"
#include"pch.h"
#include"AXRenderTargetView.cuh"

__global__ void KernelInvokeRTV()
{
	printf("Hello World\n");
	return;
}

AXRenderTargetView::AXRenderTargetView()
{
	mClearRTV = KernelInvokeRTV;

	KernelInvokeRTV << <1, 1,1>> > ();
//	KernelInvokeRTV<<<1,1>>>();
	hipDeviceSynchronize();

	mClearRTV();
	hipDeviceSynchronize();

	printf("Hello World\n");

}

AXRenderTargetView::~AXRenderTargetView()
{
}
