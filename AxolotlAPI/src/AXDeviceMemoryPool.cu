#include<pch.h>
#include"AXDeviceMemoryPool.cuh"

__device__ void* gPoolRaw = nullptr;


AXDeviceMemoryAllocator::AXDeviceMemoryAllocator(void* poolPtr, unsigned int size, unsigned int offset)
	: Size(size), Raw(poolPtr), Offset(offset)
{
	
}

AXDeviceMemoryAllocator::~AXDeviceMemoryAllocator()
{

}

AXDeviceMemoryPool::AXDeviceMemoryPool(unsigned int size)
	: mOffset(0)
{
	hipDeviceProp_t prop;
	int deviceIndex;

	hipGetDevice(&deviceIndex);
	hipGetDeviceProperties(&prop, deviceIndex);

	std::string memoryStr = std::to_string(prop.totalGlobalMem) + " bytes";
	std::string sizeStr = std::to_string(size);
	Log(std::string("Available device memory : ") + memoryStr);
	Log(std::string("Request ") + sizeStr + " bytes.");

	hipError_t error = hipMalloc(reinterpret_cast<void**>(&gPoolRaw), size);
	mRaw = gPoolRaw;

	if (error != NULL)
	{
		Log(hipGetErrorString(error));
	}

	Log(std::string("Memory requesting success, " + sizeStr + " bytes allocated."));
}

AXDeviceMemoryPool::~AXDeviceMemoryPool()
{
}

void* AXDeviceMemoryPool::Alloc(unsigned int size)
{
	void* ptr = (void*)((size_t)(mRaw) + mOffset);

	std::shared_ptr<AXDeviceMemoryAllocator> ma = std::make_shared<AXDeviceMemoryAllocator>(gPoolRaw, size, mOffset);

	mBlock.insert_or_assign(ptr , ma);

	mOffset += size;

	return ptr;
}

void AXDeviceMemoryPool::ReleasePool()
{
	hipFree(gPoolRaw);
}
