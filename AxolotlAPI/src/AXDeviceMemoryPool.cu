#include<pch.h>
#include"AXDeviceMemoryPool.cuh"

__device__ void* gPoolRaw = nullptr;


AXDeviceMemoryAllocator::AXDeviceMemoryAllocator(void* poolPtr, unsigned int size, unsigned int offset)
	: Size(size), Raw(poolPtr), Offset(offset)
{
	
}

AXDeviceMemoryAllocator::~AXDeviceMemoryAllocator()
{

}

AXDeviceMemoryPool::AXDeviceMemoryPool(size_t size)
	: mOffset(0)
{
	std::string sizeStr = std::to_string(size);
	Log(std::string("Request ") + sizeStr + " bytes.");

	hipError_t error = hipMalloc(reinterpret_cast<void**>(&gPoolRaw), size);
	mRaw = gPoolRaw;

	if (error != NULL)
	{
		Log(hipGetErrorString(error));
	}

	Log(std::string("Memory request success, " + sizeStr + " bytes allocated."));
}

AXDeviceMemoryPool::~AXDeviceMemoryPool()
{
}

void* AXDeviceMemoryPool::Alloc(unsigned int size)
{
	void* ptr = (void*)((size_t)(mRaw) + mOffset);

	std::shared_ptr<AXDeviceMemoryAllocator> ma = std::make_shared<AXDeviceMemoryAllocator>(gPoolRaw, size, mOffset);

	mBlock.insert_or_assign(ptr , ma);

	mOffset += size;

	return ptr;
}

void AXDeviceMemoryPool::ReleasePool()
{
	hipFree(gPoolRaw);
}
