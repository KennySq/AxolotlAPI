#include "hip/hip_runtime.h"
#include<pch.h>
#include"AXRasterizer.cuh"
#include"AXVertexStage.cuh"
#include"AX3DMath.h"
#include"IAXResource.h"

__device__ AXFLOAT2* gProjectedBuffer = nullptr;

AXRasterizer::AXRasterizer()
{
}

__device__ void deviceSetPixel(void* ptr, unsigned int x, unsigned int y, unsigned int width, float r, float g, float b, float a)
{
	unsigned int index = (y * width) + x;

	DWORD* texture = reinterpret_cast<DWORD*>(ptr);

	texture[index] = deviceConvertRGB(r, g, b, a);
}

__device__ void deviceDrawLine(void* ptr, AXFLOAT2 p0, AXFLOAT2 p1, unsigned int width)
{
	float deltax = p0.x - p1.x;
	float deltay = p0.y - p1.y;

	float slope = deltay / deltax;

	if (slope <= 1.0f)
	{
		AXFLOAT2 point = AXFLOAT2(p0.x, p0.y);
		for (unsigned int i = 0; i < deltax; i++)
		{
			deviceSetPixel(ptr, point.x, point.y, width, 1.0f, 0.0, 0.0f, 0.0f);
			point.x += i;
			point.y += slope;
			// update from here 2021/12/27 2:50 PM
		}
	}

	else
	{
		AXFLOAT2 point = AXFLOAT2(p0.x, p0.y);
		for (unsigned int i = 0; i < deltay; i++)
		{
			deviceSetPixel(ptr, point.x, point.y, width, 1.0f, 0.0, 0.0f, 0.0f);
			point.x += slope;
			point.y += i;
		}
	}
}

// runs triangle scale
__global__ void KernelDetermineRasterize(void* renderTarget, AXFLOAT2* projectedPoints, unsigned int width)
{
	unsigned int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	unsigned int index = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	AXFLOAT2 position0 = projectedPoints[threadIdx.x];
	AXFLOAT2 position1 = projectedPoints[threadIdx.x + 1];
	AXFLOAT2 position2 = projectedPoints[threadIdx.x + 2];

	deviceDrawLine(renderTarget, position0, position1, width);
	deviceDrawLine(renderTarget, position1, position2, width);
	deviceDrawLine(renderTarget, position2, position0, width);



}
void AXRasterizer::Process(std::shared_ptr<IAXResource> texture, unsigned int width, unsigned int height, void* projectedData, unsigned int vertexCount)
{
	// update from here 2021/12/27 11:37 AM
	dim3 grid = dim3(vertexCount / 3);
	dim3 block = dim3(32, 1, 1);
	KernelDetermineRasterize<<<grid, block>>>(texture->mRaw, projectedData, width);
	
}